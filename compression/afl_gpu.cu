#include "hip/hip_runtime.h"
#include "afl_gpu.cuh"
#include "macros.cuh"

#include <stdio.h>

template < typename T, char CWARP_SIZE >
__host__ void run_afl_compress_gpu(int bit_length, T *data, T *compressed_data, unsigned long length)
{
    int block_size = CWARP_SIZE * 8; // better occupancy 
    unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));
    afl_compress_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, data, compressed_data, length);
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_decompress_gpu(int bit_length, T *compressed_data, T *data, unsigned long length)
{
    int block_size = CWARP_SIZE * 8; // better occupancy
    unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size * CWORD_SIZE(T));
    afl_decompress_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, compressed_data, data, length);
}

template < typename T, char CWARP_SIZE >
__host__ void run_afl_decompress_value_gpu(int bit_length, T *compressed_data, T *data, unsigned long length)
{
    int block_size = CWARP_SIZE * 8; // better occupancy
    unsigned long block_number = (length + block_size * CWORD_SIZE(T) - 1) / (block_size);
    afl_decompress_value_gpu <T, CWARP_SIZE> <<<block_number, block_size>>> (bit_length, compressed_data, data, length);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_compress_gpu (int bit_length, T *data, T *compressed_data, unsigned long length)
{
    unsigned int warp_lane = (threadIdx.x % CWARP_SIZE); 
    unsigned long data_block = blockIdx.x * blockDim.x + threadIdx.x - warp_lane;
    unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
    unsigned long cdata_id = data_block * bit_length + warp_lane;

    afl_compress_base_gpu <T, CWARP_SIZE> (bit_length, data_id, cdata_id, data, compressed_data, length);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_decompress_gpu (int bit_length, T *compressed_data, T * decompress_data, unsigned long length)
{
    unsigned int warp_lane = (threadIdx.x % CWARP_SIZE); 
    unsigned long data_block = blockIdx.x * blockDim.x + threadIdx.x - warp_lane;
    unsigned long data_id = data_block * CWORD_SIZE(T) + warp_lane;
    unsigned long cdata_id = data_block * bit_length + warp_lane;

    afl_decompress_base_gpu <T, CWARP_SIZE> (bit_length, cdata_id, data_id, compressed_data, decompress_data, length);
}

template < typename T, char CWARP_SIZE >
__global__ void afl_decompress_value_gpu (int bit_length, T *compressed_data, T * decompress_data, unsigned long length)
{
    unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length)
    {
        decompress_data[tid] = afl_decompress_base_value_gpu <T, CWARP_SIZE> (bit_length, compressed_data, tid);
    }
}


template <typename T, char CWARP_SIZE>
__device__  __host__ void afl_compress_base_gpu (int bit_length, unsigned long data_id, unsigned long comp_data_id, T *data, T *compressed_data, unsigned long length)
{
    T v1, value = 0;
    unsigned int v1_pos=0, v1_len;
    unsigned long pos=comp_data_id, pos_data=data_id;

    for (unsigned int i = 0; i < CWORD_SIZE(T) && pos_data < length; ++i) 
    {
        v1 = data[pos_data];
        pos_data += CWARP_SIZE;

        if (v1_pos >= CWORD_SIZE(T) - bit_length){
            v1_len = CWORD_SIZE(T) - v1_pos;
            value = value | (GETNBITS(v1, v1_len) << v1_pos);

            compressed_data[pos] = value;

            v1_pos = bit_length - v1_len;
            value = GETNPBITS(v1, v1_pos, v1_len); 

            pos += CWARP_SIZE;  
        } else {
            v1_len = bit_length;
            value = value | (GETNBITS(v1, v1_len) << v1_pos);
            v1_pos += v1_len;
        }
    }
    if (pos_data >= length  && pos_data < length + CWARP_SIZE)
    {
        compressed_data[pos] = value;
    }
}

template <typename T, char CWARP_SIZE>
__device__ __host__ void afl_decompress_base_gpu (int bit_length, unsigned long comp_data_id, unsigned long data_id, T *compressed_data, T *data, unsigned long length)
{
    unsigned long pos = comp_data_id, pos_decomp = data_id;
    unsigned int v1_pos = 0, v1_len;
    T v1, ret;

    if (pos_decomp > length ) // Decompress not more elements then length
        return;
    v1 = compressed_data[pos];
    for (unsigned int i = 0; i < CWORD_SIZE(T) && pos_decomp < length; ++i)
    {
        if (v1_pos >= CWORD_SIZE(T) - bit_length){ 
            v1_len = CWORD_SIZE(T) - v1_pos;
            ret = GETNPBITS(v1, v1_len, v1_pos);

            pos += CWARP_SIZE;  
            v1 = compressed_data[pos];

            v1_pos = bit_length - v1_len;
            ret = ret | ((GETNBITS(v1, v1_pos))<< v1_len);
        } else {
            v1_len = bit_length;
            ret = GETNPBITS(v1, v1_len, v1_pos);
            v1_pos += v1_len;
        }

        data[pos_decomp] = ret;
        pos_decomp += CWARP_SIZE;
    }
}

template <typename T, char CWARP_SIZE>
__device__ __host__ T afl_decompress_base_value_gpu (
        int bit_length, 
        T *compressed_data, 
        unsigned long pos
        )
{
    int data_block = pos / (CWARP_SIZE * CWORD_SIZE(T));
    int pos_in_block = (pos % (CWARP_SIZE * CWORD_SIZE(T)));
    int pos_in_warp_lane = pos_in_block % CWARP_SIZE;
    int pos_in_warp_comp_block = pos_in_block / CWARP_SIZE;


    unsigned long cblock_id = data_block * ( CWARP_SIZE * bit_length)
        + pos_in_warp_lane 
        + ((pos_in_warp_comp_block * bit_length) / CWORD_SIZE(T)) * CWARP_SIZE;

    /*printf("data_block, %d, pos_in_block, %d, pos_in_warp_lane, %d, pos_in_warp_comp_block, %d, cblock_id, %ld %ld %ld\n", data_block, pos_in_block, pos_in_warp_lane, pos_in_warp_comp_block, cblock_id,*/
            /*(long int)data_block * ( CWARP_SIZE * bit_length), (long int)((pos_in_warp_comp_block * bit_length)/CWORD_SIZE(T)));*/


    int bit_pos = pos_in_warp_comp_block * bit_length % CWORD_SIZE(T);
    int bit_ret = bit_pos <= CWORD_SIZE(T)  - bit_length  ? bit_length : CWORD_SIZE(T) - bit_pos;

    T ret = GETNPBITS(compressed_data[cblock_id], bit_ret, bit_pos);

    if (bit_ret < bit_length)
        ret |= GETNBITS(compressed_data[cblock_id+CWARP_SIZE], bit_length - bit_ret) << bit_ret;

    return ret;
}

// For now only those versions are available and will be compiled and linked
// This is intentional !!
#define GFL_SPEC(X, A) \
    template __device__ __host__ void afl_decompress_base_gpu <X, A> (int, unsigned long comp_data_id, unsigned long data_id, X *compressed_data, X *data, unsigned long length);\
    template __device__ __host__ void afl_compress_base_gpu <X, A> (int, unsigned long, unsigned long, X *, X *, unsigned long );\
    template __device__ __host__ X afl_decompress_base_value_gpu <X, A> ( int bit_length, X *compressed_data, unsigned long pos);\
    template __global__ void afl_decompress_gpu <X, A> ( int bit_length, X *compressed_data, X * decompress_data, unsigned long length);\
    template __global__ void afl_compress_gpu < X, A> ( int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __global__ void afl_decompress_value_gpu <X, A> (int bit_length, X *compressed_data, X * decompress_data, unsigned long length);\
    template __host__ void run_afl_compress_gpu <X, A> (int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __host__ void run_afl_decompress_gpu <X, A> (int bit_length, X *data, X *compressed_data, unsigned long length);\
    template __host__ void run_afl_decompress_value_gpu <X, A> (int bit_length, X *compressed_data, X *data, unsigned long length);

// A fast aligned version WARP_SIZE = 32
#define AFL_SPEC(X) GFL_SPEC(X, 32)
FOR_EACH(AFL_SPEC, int, long, unsigned int, unsigned long)

// Non aligned version - identical to classical CPU/GPU version (up to 10x slower then AFL)
#define FL_SPEC(X) GFL_SPEC(X, 1)
FOR_EACH(FL_SPEC, int, long, unsigned int, unsigned long)
