//#define CATCH_CONFIG_MAIN  // This tells Catch to provide a main() - only do this in one cpp file
#define CATCH_CONFIG_RUNNER
#include "catch.hpp"
#include <stdio.h>
#include <stdlib.h>
#include "tools/macros.cuh"

/* struct OtherOpt { */
/*     OtherOpt() : deviceNumber(0), showHelp(false) {} */

/*     std::string processName; */
/*     int deviceNumber; */
/*     bool showHelp; */
    
/*     void setValidDeviceNumber( int i ) { */
/*         int deviceCount = 0; */
/*         hipGetDeviceCount(&deviceCount); */
/*         if( i < 0 || i > deviceCount ) { */
/*             Catch::cout()<<"The device number is incorrect, please set valid cuda device number\n"; */
/*             exit(0); */
/*         } */
/*         deviceNumber = i; */

/*         hipSetDevice(deviceNumber); */

/*         hipDeviceProp_t deviceProp; */
/*         hipGetDeviceProperties(&deviceProp, deviceNumber); */

/*         Catch::cout() <<"Device "<< deviceNumber <<": "<<deviceProp.name<<"\n"; */
/*     } */
/* }; */

void setValidDeviceNumber( int i ) {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if( i < 0 || i > deviceCount ) {
        Catch::cout()<<"The device number is incorrect, please set valid cuda device number\n";
        exit(0);
    }
    int deviceNumber = i;

    hipSetDevice(deviceNumber);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceNumber);

    if (if_debug()) {
        Catch::cout() <<"Device "<< deviceNumber <<": "<<deviceProp.name<<"\n";
    }
}

int main(int argc, char** argv)
{

    Catch::Session session;
    session.applyCommandLine(argc, argv, Catch::Session::OnUnusedOptions::Ignore);

    char* GPU_DEVICE;
    GPU_DEVICE = getenv ("GPU_DEVICE");
    int dev_id = 0;
    if (GPU_DEVICE != NULL)
        dev_id = atoi(GPU_DEVICE);

    if (dev_id >= 0) 
        setValidDeviceNumber(dev_id);
        

    /* OtherOpt config; */
    /* Catch::Clara::CommandLine<OtherOpt> cli; */

    /* cli["-D"]["--device"] */
    /*     .describe( "Set cuda device" ) */
    /*     .bind( &OtherOpt::setValidDeviceNumber, "deviceNumber"); */

    /* cli.parseInto( argc-1, argv+1, config );  //parse extra args (like cuda device) */

    /* if(session.configData().showHelp) { */
    /*     Catch::cout() << "\ngpu_compression_lib specific options\n"; */
    /*     cli.usage(Catch::cout(), session.configData().processName); */
    /* } */
    
    return session.run();
}
