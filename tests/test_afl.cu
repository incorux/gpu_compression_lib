#include "catch.hpp"

#include "tools/tools.cuh"

#include "compression/afl_gpu.cuh"
#include "compression/pafl_gpu.cuh"

#include <fstream>
#include <string>
#include <map>

#define PPRINT_THROUGPUT(name, data_size) {printf("%c[1;34m",27);  printf name; printf("%c[30m, %c[37m", 27,27); TIMEIT_PRINT_THROUGPUT(data_size);}

template <typename T, int CWARP_SIZE>
class test_afl
{
public:
    void allocateMemory() {
        mmCudaMallocHost(manager, (void**)&host_data,  data_size);
        mmCudaMallocHost(manager, (void**)&host_data2, data_size);

        mmCudaMalloc(manager, (void **) &dev_out, compressed_data_size); 
        mmCudaMalloc(manager, (void **) &dev_data, data_size);
    }

    void initializeData(int bit_length) {
        big_random_block(max_size, bit_length, host_data);
    }

    void transferDataToGPU() {
        gpuErrchk( hipMemcpy(dev_data, host_data, data_size, hipMemcpyHostToDevice) );
    }

    void cleanBeforeCompress() {
        hipMemset(dev_out, 0, compressed_data_size); // Clean up before compression
    }

    virtual void compressData(int bit_length) {
        run_afl_compress_gpu <T, CWARP_SIZE> (bit_length, dev_data, dev_out, max_size);
    }

    void errorCheck() { 
        cudaErrorCheck();
    }

    void cleanBeforeDecompress() {
        hipMemset(dev_data, 0, data_size); // Clean up before decompression
    }

    virtual void decompressData(int bit_length) {
        run_afl_decompress_gpu <T, CWARP_SIZE> (bit_length, dev_out, dev_data, max_size);
    }

    void transferDataFromGPU() {
        hipMemset(host_data2, 0, data_size); 
        gpuErrchk(hipMemcpy(host_data2, dev_data, data_size, hipMemcpyDeviceToHost));
    }

    void run(unsigned int max_size, bool print = false)
    {
        this->max_size = max_size;
        cword = sizeof(T) * 8;
        data_size = max_size * sizeof(T);
        // for size less then cword we actually will need more space than original data
        compressed_data_size = (max_size < cword  ? cword : max_size) * sizeof(T);

        allocateMemory();
        TIMEIT_SETUP();

        for (unsigned int i = 1; i < cword; ++i) {
            initializeData(i);

            TIMEIT_START();
            transferDataToGPU();
            TIMEIT_END("M->G");
            
            cleanBeforeCompress();
            
            TIMEIT_START();
            compressData(i);
            TIMEIT_END("*comp");
            
            errorCheck();

            cleanBeforeDecompress();

            TIMEIT_START();
            decompressData(i);
            TIMEIT_END("*comp");
            
            errorCheck();

            TIMEIT_START();
            transferDataFromGPU();
            TIMEIT_END("G->M");

            CHECK(testData()==0);
            
            if(print) PPRINT_THROUGPUT(("%s fl=%d", __PRETTY_FUNCTION__, i), data_size);
        }
    }

    T testData() {
       return compare_arrays(host_data2, host_data, max_size);
    }

    virtual ~test_afl () {
        mmCudaFreeAll(manager);
    }

protected:
    T *dev_out;
    T *dev_data;
    T *host_data; 
    T *host_data2;

    int cword;

    int compressed_data_size;
    unsigned long data_size;
    unsigned long max_size;

    mmManager manager;
};


template <typename T, int CWARP_SIZE> class test_afl_random_access: public test_afl<T, CWARP_SIZE> {
public: virtual void decompressData(int bit_length) {
        run_afl_decompress_value_gpu <T, CWARP_SIZE> (bit_length, this->dev_out, this->dev_data, this->max_size);
    }
};

#define SMALL_DATA_SET 1000 
#define MEDIUM_DATA_SET  100000
#define PERF_DATA_SET  100000000

#define RUN_TEST(NAME, CNAME, PARAM)\
TEST_CASE( NAME " test set", "[" NAME "]" ) {\
    SECTION("int: SMALL data set")   {CNAME <int, PARAM> ().run(SMALL_DATA_SET);}\
    SECTION("int: MEDIUM data set")  {CNAME <int, PARAM>  ().run(MEDIUM_DATA_SET);}\
    SECTION("long: SMALL data set")  {CNAME <long, PARAM> ().run(SMALL_DATA_SET);}\
    SECTION("long: MEDIUM data set")  {CNAME <long, PARAM> ().run(MEDIUM_DATA_SET);}\
}

RUN_TEST("AFL", test_afl, 32);
RUN_TEST("FL", test_afl, 1);

RUN_TEST("RAFL", test_afl_random_access, 32);
RUN_TEST("RFL", test_afl_random_access, 1);

#define RUN_PERF_TEST(NAME, CNAME, PARAM)\
TEST_CASE( NAME " performance test", "[" NAME "][PERF][hide]" ) {\
    SECTION("int: PERF data set")   {CNAME <int, PARAM> ().run(PERF_DATA_SET, true);}\
    SECTION("long: PERF data set")  {CNAME <int, PARAM>  ().run(PERF_DATA_SET, true);}\
}

RUN_PERF_TEST("AFL", test_afl, 32);
RUN_PERF_TEST("FL", test_afl, 1);

RUN_PERF_TEST("RAFL", test_afl_random_access, 32);
RUN_PERF_TEST("RFL", test_afl_random_access, 1);


