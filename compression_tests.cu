#include "hip/hip_runtime.h"
#include "compression/tools.cuh"
#include "compression/avar_gpu.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define PPRINT(name) printf("%c[1;34m",27);  printf name; printf("%c[30m Status: %c[1;32mCORRECT%c[37m ", 27,27,27); TIMEIT_PRINT();
/*#define PPRINT(name) printf("%c[1;34m",27);  printf name; printf("%c[30m Status: %c[1;32mOK%c[37m \n", 27,27,27);*/
#define PPRINT_MANY(name) printf("%c[1;34m",27);  printf name; printf("%c[30m: %c[1;32mOK%c[37m ", 27,27,27);

#define PPRINT_THROUGPUT(name, data_size) printf("%c[1;34m",27);  printf name; printf("%c[30m, %c[1;32mOK%c[37m, ", 27,27,27); TIMEIT_PRINT_THROUGPUT(data_size);

void avar_gpu_test(size_t max_size)
{
    int *dev_out;
    int *dev_data;
    int *host_data, *host_data2;

    mmManager manager;

    TIMEIT_SETUP();

    mmCudaMallocHost(manager,(void**)&host_data, max_size * sizeof(int));
    mmCudaMallocHost(manager,(void**)&host_data2, max_size * sizeof(int));

    mmCudaMalloc(manager, (void **) &dev_out, max_size * sizeof(int)); // maximal compression size
    mmCudaMalloc(manager, (void **) &dev_data, max_size * sizeof(int));

    for (unsigned int i = 2; i <= 31; ++i) {
        big_random_block(max_size, pow((double)2,(double)(i-1))-1, host_data);

        TIMEIT_START();
        gpuErrchk( hipMemcpy(dev_data, host_data, max_size * sizeof(int), hipMemcpyHostToDevice) );
        TIMEIT_END("M->G");

        avar_header comp_h = { i };
        hipMemset(dev_out, 0, max_size * sizeof(int)); // Clean up before compression

        TIMEIT_START();
        run_avar_compress_gpu(comp_h, dev_data, dev_out, max_size);
        TIMEIT_END("*comp");
        cudaErrorCheck();

        hipMemset(dev_data, 0, max_size * sizeof(int)); // Clean up before decompression

        TIMEIT_START();
        run_avar_decompress_gpu(comp_h, dev_out, dev_data, max_size);
        TIMEIT_END("*decomp");
        cudaErrorCheck();

        hipMemset(host_data2, 0, max_size * sizeof(int)); // Clean up before compression
        TIMEIT_START();
        gpuErrchk(hipMemcpy(host_data2, dev_data, max_size * sizeof(int), hipMemcpyDeviceToHost));
        TIMEIT_END("G->M");

        compare_arrays(host_data2, host_data, max_size);
        PPRINT_THROUGPUT(("GPU avar%d", i), max_size * sizeof(int));

    }

    mmCudaFreeAll(manager);
}

int main(int argc, char *argv[])
{
    size_t max_size = 100000000;

    if (argc > 1 && atol(argv[1]))
        max_size = atol(argv[1]);

    printf("Data size: %ld\n", max_size );
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        avar_gpu_test(max_size);
    }
    return 0;
}
/* vim: set fdm=syntax: */
