#include "compression/tools.cuh"
#include "compression/avar_gpu.cuh"
#include "compression/pavar_gpu.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define PPRINT(name) printf("%c[1;34m",27);  printf name; printf("%c[30m Status: %c[1;32mCORRECT%c[37m ", 27,27,27); TIMEIT_PRINT();
/*#define PPRINT(name) printf("%c[1;34m",27);  printf name; printf("%c[30m Status: %c[1;32mOK%c[37m \n", 27,27,27);*/
#define PPRINT_MANY(name) printf("%c[1;34m",27);  printf name; printf("%c[30m: %c[1;32mOK%c[37m ", 27,27,27);

#define PPRINT_THROUGPUT(name, data_size) printf("%c[1;34m",27);  printf name; printf("%c[30m, %c[1;32mOK%c[37m, ", 27,27,27); TIMEIT_PRINT_THROUGPUT(data_size);

void avar_gpu_test(unsigned long max_size)
{
    int *dev_out;
    int *dev_data;
    int *host_data, *host_data2;

    // for size less then 32 we actually will need more space than original data
    int compressed_data_size = (max_size < 32 ? 32 : max_size) * sizeof(int); 

    int data_size = max_size * sizeof(int); 

    mmManager manager;

    TIMEIT_SETUP();

    mmCudaMallocHost(manager, (void**)&host_data,  data_size);
    mmCudaMallocHost(manager, (void**)&host_data2, data_size);

    mmCudaMalloc(manager, (void **) &dev_out, compressed_data_size); 
    mmCudaMalloc(manager, (void **) &dev_data, data_size);

    for (unsigned int i = 2; i <= 31; ++i) {
        big_random_block(max_size, i, host_data);

        TIMEIT_START();
        gpuErrchk( hipMemcpy(dev_data, host_data, data_size, hipMemcpyHostToDevice) );
        TIMEIT_END("M->G");

        avar_header comp_h = { i };
        hipMemset(dev_out, 0, compressed_data_size); // Clean up before compression

        TIMEIT_START();
        run_avar_compress_gpu(comp_h, dev_data, dev_out, max_size);
        TIMEIT_END("*comp");
        cudaErrorCheck();

        hipMemset(dev_data, 0, data_size); // Clean up before decompression

        TIMEIT_START();
        run_avar_decompress_gpu(comp_h, dev_out, dev_data, max_size);
        TIMEIT_END("*decomp");
        cudaErrorCheck();

        hipMemset(host_data2, 0, data_size); 
        TIMEIT_START();
        gpuErrchk(hipMemcpy(host_data2, dev_data, data_size, hipMemcpyDeviceToHost));
        TIMEIT_END("G->M");

        compare_arrays(host_data2, host_data, max_size);

        PPRINT_THROUGPUT(("GPU avar%d", i), data_size);
    }

    mmCudaFreeAll(manager);
}

void pavar_gpu_test(unsigned long max_size)
{
    int *dev_out;
    int *dev_data;
    int *host_data, *host_data2;
    int *dev_data_patch_index, *dev_data_patch_values, *dev_data_patch_count;
    int *dev_queue_patch_index, *dev_queue_patch_values, *dev_queue_patch_count;
    int outlier_count = 0.2 * max_size;

    mmManager manager;
    TIMEIT_SETUP();

    mmCudaMallocHost(manager,(void**)&host_data, max_size * sizeof(int));
    mmCudaMallocHost(manager,(void**)&host_data2, max_size * sizeof(int));

    mmCudaMalloc(manager, (void **) &dev_out, max_size * sizeof(int)); // maximal compression size
    mmCudaMalloc(manager, (void **) &dev_data, max_size * sizeof(int));
    
    mmCudaMalloc(manager, (void **) &dev_data_patch_count, sizeof(int));
    mmCudaMalloc(manager, (void **) &dev_data_patch_index, outlier_count * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev_data_patch_values, outlier_count * sizeof(int));

    mmCudaMalloc(manager, (void **) &dev_queue_patch_count, sizeof(int));
    mmCudaMalloc(manager, (void **) &dev_queue_patch_index, outlier_count * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev_queue_patch_values, outlier_count * sizeof(int));

    for (unsigned int i = 2; i <= 31; ++i) {
        big_random_block_with_outliers(max_size, outlier_count, i, i + 3, host_data);

        TIMEIT_START();
        gpuErrchk( hipMemcpy(dev_data, host_data, max_size * sizeof(int), hipMemcpyHostToDevice) );
        TIMEIT_END("M->G");

        pavar_header comp_h = { i, 4 };
        hipMemset(dev_out, 0, max_size * sizeof(int)); // Clean up before compression
        hipMemset(dev_data_patch_count, 0, sizeof(int)); // Clean up before compression
        hipMemset(dev_queue_patch_count, 0, sizeof(int)); // Clean up before compression

        TIMEIT_START();
        //TODO
        run_pavar_compress_gpu(
                comp_h, 
                dev_data, 
                dev_out, 
                max_size,
                
                dev_queue_patch_values, 
                dev_queue_patch_index,
                dev_queue_patch_count,

                dev_data_patch_values,
                dev_data_patch_index,
                dev_data_patch_count
                );

        TIMEIT_END("*comp");
        cudaErrorCheck();

        hipMemset(dev_data, 0, max_size * sizeof(int)); // Clean up before decompression

        TIMEIT_START();
        //TODO
        /*run_pavar_decompress_gpu(comp_h, dev_out, dev_data, max_size);*/
        TIMEIT_END("*decomp");
        cudaErrorCheck();

        /*hipMemset(host_data2, 0, max_size * sizeof(int)); */
        TIMEIT_START();
        /*gpuErrchk(hipMemcpy(host_data2, dev_data, max_size * sizeof(int), hipMemcpyDeviceToHost));*/
        TIMEIT_END("G->M");

        /*compare_arrays(host_data2, host_data, max_size);*/

        PPRINT_THROUGPUT(("GPU pavar%d", i), max_size * sizeof(int));
    }

    mmCudaFreeAll(manager);
}

int main(int argc, char *argv[])
{
    unsigned long max_size = 100000000;

    if (argc > 1 && atol(argv[1]))
        max_size = atol(argv[1]);

    printf("Data size: %ld\n", max_size );
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        avar_gpu_test(max_size);
        /*pavar_gpu_test(max_size);*/
    }
    return 0;
}
/* vim: set fdm=syntax: */
