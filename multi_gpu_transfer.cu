#include "hip/hip_runtime.h"
#include "compression/tools.cuh"
#include "compression/avar_gpu.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

int gpuid_0=1, gpuid_1=2;

#define PPRINT_THROUGPUT(name, data_size) printf("%c[1;34m",27);  printf name; printf("%c[30m,%c[37m ", 27,27); TIMEIT_PRINT_THROUGPUT(data_size);

__global__ void saxpy(size_t n, int a, int *x, int *y)
{    
    // Determine element to process from thread index    
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < n; tid += blockDim.x * gridDim.x) 
        y[tid] += a*x[tid];
}

void multi_gpu_compress(size_t max_size, unsigned int bit_length, bool direct_copy)
{
    mmManager manager;
    int *dev0_data, *dev0_comp_out;
    int *dev1_data, *dev1_data_out, *dev1_comp_out;

    long comp_size = ((max_size * bit_length)/32 +32) * sizeof(int);


    gpuErrchk(hipSetDevice(gpuid_0));
    mmCudaMalloc(manager, (void **) &dev0_data, max_size * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev0_comp_out, comp_size);

    gpuErrchk(hipSetDevice(gpuid_1));
    mmCudaMalloc(manager, (void **) &dev1_data, max_size * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev1_data_out, max_size * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev1_comp_out, comp_size);

    gpuErrchk(hipSetDevice(gpuid_0));
    avar_header comp_h = { bit_length } ;

    TIMEIT_SETUP();

    TIMEIT_START();
    run_avar_compress_gpu(comp_h, dev0_data, dev0_comp_out, max_size);
    TIMEIT_END("*C");
    cudaErrorCheck();

    int *dev_data_source = dev0_comp_out;
    gpuErrchk(hipSetDevice(gpuid_1));

    if (direct_copy)
    {
        TIMEIT_START();
        hipMemcpyPeerAsync(dev1_comp_out, gpuid_1, dev0_comp_out, gpuid_0, comp_size);
        hipDeviceSynchronize();
        TIMEIT_END("*copy");
        dev_data_source = dev1_comp_out;
        cudaErrorCheck();
    }

    TIMEIT_START();
    run_avar_decompress_gpu(comp_h, dev_data_source, dev1_data, max_size);
    TIMEIT_END("*D");
    cudaErrorCheck();
    
    TIMEIT_START();
    saxpy <<<4096, 512>>> (max_size, 10, dev1_data, dev1_data_out);
    cudaErrorCheck();
    TIMEIT_END("saxpy");
    
    PPRINT_THROUGPUT(("MGPU%s compr avar%d", direct_copy ? "copy":"access", bit_length), max_size * sizeof(int));

    mmCudaFreeAll(manager);
}


void multi_gpu(size_t max_size, bool direct_copy)
{
    mmManager manager;
    int *dev0_data, *dev1_data;
    int *dev1_data_out;

    gpuErrchk(hipSetDevice(gpuid_0));
    mmCudaMalloc(manager, (void **) &dev0_data, max_size * sizeof(int));

    gpuErrchk(hipSetDevice(gpuid_1));
    mmCudaMalloc(manager, (void **) &dev1_data, max_size * sizeof(int));
    mmCudaMalloc(manager, (void **) &dev1_data_out, max_size * sizeof(int));

    TIMEIT_SETUP();

    int *dev_data_source = dev0_data;

    if (direct_copy)
    {
        TIMEIT_START();
        /*hipMemcpy(dev1_data, dev0_data, max_size * sizeof(int), hipMemcpyDefault);*/
        hipMemcpyPeerAsync(dev1_data, gpuid_1, dev0_data, gpuid_0, max_size * sizeof(int));
        hipDeviceSynchronize();
        TIMEIT_END("*copy");
        dev_data_source = dev1_data;
    }
    
    TIMEIT_START();
    saxpy <<<4096, 512>>> (max_size, 10, dev_data_source, dev1_data_out);
    cudaErrorCheck();
    TIMEIT_END("saxpy");
    
    PPRINT_THROUGPUT(("MGPU%s", direct_copy ? "copy":"access"), max_size * sizeof(int));

    mmCudaFreeAll(manager);
}

int main(int argc, char *argv[])
{

    size_t max_size = 10000000;
    printf("%s [size] [dev0_id, dev1_id]\n", argv[0]);
    if(argc > 1) {
        if ( atol(argv[1]))
            max_size = atol(argv[1]);

        if (argc == 4) {
            gpuid_0 = atoi(argv[2]);
            gpuid_1 = atoi(argv[3]);
        }
    }

    printf("Data size: %ld,using device %d and device %d\n", max_size, gpuid_0, gpuid_1 );


    int can_access_peer_0_1, can_access_peer_1_0;
    gpuErrchk(hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid_0, gpuid_1));
    gpuErrchk(hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid_1, gpuid_0));
    printf("can acces device 0->1: %d  1->0 %d\n",can_access_peer_0_1, can_access_peer_1_0 );

    gpuErrchk(hipSetDevice(gpuid_0));
    gpuErrchk(hipDeviceEnablePeerAccess(gpuid_1, 0));

    gpuErrchk(hipSetDevice(gpuid_1));
    gpuErrchk(hipDeviceEnablePeerAccess(gpuid_0, 0));


    multi_gpu(max_size, true);
    multi_gpu(max_size, false);

    for (int i = 2; i < 32; ++i)
    {
        multi_gpu_compress(max_size, i, true);
        multi_gpu_compress(max_size, i, false);
    }
    
    return 0;
}
